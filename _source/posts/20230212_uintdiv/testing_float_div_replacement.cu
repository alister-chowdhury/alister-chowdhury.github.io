

// Did apparently break my laptop...


// Theoretically *should* work up until 16777216



/*
uint fastUintDiv(uint a, uint b)
{
    // return uint(float(a) / float(b));                            // requires explicit fdiv, not rcp
    // return uint(asfloat(1 + asuint(float(a) * rcp(float(b)))));  // 36 cycles, works past 0xffff
    return uint(float(a) * rcp(float(b)) + asfloat(0x38000001));    // 32 cycles, works if x, y <= 65174
}





#define asfloat  uintBitsToFloat
#define asuint   floatBitsToUint
#define rcp(x)   (1./x)



// INTEL: (a <= 3981553, b <= 111602) 
// NVIDIA: (a <= VERY LARGE,  b <= 988)
uint fastUintDiv(uint a, uint b)
{
    return uint(asfloat(1u + asuint(float(a) * rcp(float(b)))));  // 36 cycles, works past 0xffff
}



uint fastUintDiv2(uint a, uint b)
{
    return uint(asfloat(2u + asuint(float(a) * rcp(float(b)))));  // 36 cycles, works past 0xffff
}








uint fasterUintDiv(uint a, uint b)
{
    return uint(float(a) * rcp(float(b)) + asfloat(0x38000001u));    // 32 cycles, works if x, y <= 65174
}




*/




/*

WEBGL ES


int fdiv(int a, int b)
{
    highp float x = float(a);
    highp float y = float(b);
    highp float z = (x ) / y;
    return int(z);
}




void mainImage( out vec4 fragColor, in vec2 fragCoord )
{
    
    ivec2 fc = ivec2(fragCoord ) + 1;
    //fc.x = iFrame;
    int f0 = fdiv(fc.x, fc.y);
    int f1 = fc.x / fc.y;
    
    float eq = float(f0 == f1);
    float lt = float(f0 < f1);
    float gt = float(f0 > f1);


    // Output to screen
    fragColor = vec4(vec3(lt, eq, gt),1.0);
    
    fragColor.xyz = vec3(eq);
    
}
*/


__global__ void t(int offsetx, int offsety, int* allocator, int* writeback)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x + 1 + offsetx;
    int y = blockDim.y * blockIdx.y + threadIdx.y + 1 + offsety;
    int z0 = x / y;
    int z1 = int(float(x) / float(y));
    if(z0 != z1)
    {
        int k = atomicAdd(allocator, 1);
        // writeback[k*2 + 0] = x;
        // writeback[k*2 + 1] = y;
    }
}




#include <hip/hip_runtime.h>
#include <stdio.h>


// Have run to 8781824, and apparently no breakage.


int main(void)
{
    int* allocator;
    int* writeback;
    
    hipMalloc(&allocator, sizeof(int));
    hipMemset(allocator, 0, sizeof(int));
    hipMalloc(&writeback, sizeof(int) * 1024 * 1024);

    dim3 block(32, 32);
    dim3 grid(1024, 1024);

    dim3 periteration (block.x * grid.x, block.y * grid.y);

    // for(int x=0; x<131072; ++x)
    // for(int y=0; y<131072; ++y)
    for(int x=0; x<131072; ++x)
    for(int y=0; y<=x; ++y)
    {
        int offsetx = periteration.x * x;
        int offsety = periteration.y * y;
        t<<<grid, block>>>(offsetx, offsety, allocator, writeback);

        int numNotOk;
        hipMemcpy(&numNotOk, allocator, sizeof(int), hipMemcpyDeviceToHost);


        printf("[%i, %i] => [%i, %i], num-invalid = %i\n", offsetx, offsety, offsetx + periteration.x, offsety + periteration.y, numNotOk);
        if(numNotOk)
        {
            goto escape;
        }

    }

    escape:
    return 0;

}